#include "hip/hip_runtime.h"
// Copyright (c) 2025 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// The file has been adapted from DeepSeek DeepEP project
// Copyright (c) 2025 DeepSeek
// Licensed under the MIT License -
// https://github.com/deepseek-ai/DeepEP/blob/main/LICENSE

// clang-format off
#include <nvshmem.h>
#include <nvshmemx.h>
#include <infiniband/mlx5dv.h>
#include <non_abi/device/threadgroup/nvshmemi_common_device_defines.cuh>
#include <device_host_transport/nvshmem_common_ibgda.h>
// clang-format on

#include "paddle/fluid/distributed/collective/deep_ep/kernels/configs.cuh"
#include "paddle/fluid/distributed/collective/deep_ep/kernels/exception.cuh"
#include "paddle/fluid/distributed/collective/deep_ep/kernels/ibgda_device.cuh"
#include "paddle/fluid/distributed/collective/deep_ep/kernels/launch.cuh"

namespace deep_ep {

namespace internode_ll {

__global__ void barrier_all() { nvshmemx_barrier_all_block(); }

void barrier_all(hipStream_t stream) {
  constexpr int kNumThreads = 1;

  SETUP_LAUNCH_CONFIG(1, kNumThreads, stream);
  LAUNCH_KERNEL(&cfg, barrier_all);
}

template <int kNumThreads>
__launch_bounds__(kNumThreads, 1) __global__ void clean_low_latency_buffer(
    int* clean_0, int num_clean_int_0, int* clean_1, int num_clean_int_1) {
  // Barrier before cleaning (in case of unfinished chunked EP)
  nvshmemx_barrier_all_block();

  // Clean
  auto thread_id = static_cast<int>(threadIdx.x);
#pragma unroll
  for (int i = thread_id; i < num_clean_int_0; i += kNumThreads) clean_0[i] = 0;
#pragma unroll
  for (int i = thread_id; i < num_clean_int_1; i += kNumThreads) clean_1[i] = 0;

  // Barrier after cleaning (make sure low-latency mode work fine)
  nvshmemx_barrier_all_block();
}

void clean_low_latency_buffer(int* clean_0,
                              int num_clean_int_0,
                              int* clean_1,
                              int num_clean_int_1,
                              hipStream_t stream) {
  constexpr int kNumThreads = 256;

  SETUP_LAUNCH_CONFIG(1, kNumThreads, stream);
  LAUNCH_KERNEL(&cfg,
                clean_low_latency_buffer<kNumThreads>,
                clean_0,
                num_clean_int_0,
                clean_1,
                num_clean_int_1);
}

template <bool kUseFP8, int kNumWarpGroups, int kNumWarpsPerGroup, int kHidden>
__global__ __launch_bounds__(
    kNumWarpGroups* kNumWarpsPerGroup * 32,
    1) void dispatch(void* packed_recv_x,
                     float* packed_recv_x_scales,
                     int* packed_recv_src_info,
                     int64_t* packed_recv_layout_range,
                     int* packed_recv_count,
                     void* rdma_recv_x,
                     int* rdma_recv_count,
                     void* rdma_x,
                     const void* x,
                     const int64_t* topk_idx,
                     int* atomic_counter_per_expert,
                     int* atomic_finish_counter_per_expert,
                     int* next_clean,
                     int num_next_clean_int,
                     int num_tokens,
                     int num_max_dispatch_tokens_per_rank,
                     int num_topk,
                     int num_experts,
                     int rank,
                     int num_ranks,
                     int phases) {
  const auto sm_id = static_cast<int>(blockIdx.x);
  const auto thread_id = static_cast<int>(threadIdx.x);
  const auto warp_id = thread_id / 32, lane_id = get_lane_id();
  const auto num_sms = static_cast<int>(gridDim.x);
  const auto num_warps = kNumWarpGroups * kNumWarpsPerGroup;
  const auto num_local_experts = num_experts / num_ranks;
  const auto warp_group_id = warp_id / kNumWarpsPerGroup;
  const auto sub_warp_id = warp_id % kNumWarpsPerGroup;
  const auto responsible_expert_idx = sm_id * kNumWarpGroups + warp_group_id;

  // FP8 staffs
  constexpr int kNumPerChannels = 128;
  constexpr float kFP8Margin = 1e-4, kFP8Amax = 448,
                  kFP8AmaxInv = 1.0f / 448.0f;
  const int num_scales = kHidden / kNumPerChannels;
  const size_t hidden_bytes =
      kHidden * (kUseFP8 ? sizeof(__hip_fp8_storage_t) : sizeof(hip_bfloat16));
  const size_t hidden_int4 = hidden_bytes / sizeof(int4);

  // Message package: hidden data, FP8 scales, index at source
  // NOTES: currently we have 3 reserved int fields for future use
  using vec_t = typename std::conditional<kUseFP8, int2, int4>::type;
  const size_t num_bytes_per_msg =
      sizeof(int4) + (kUseFP8 ? (kHidden + num_scales * sizeof(float))
                              : (kHidden * sizeof(hip_bfloat16)));
  const size_t num_int4_per_msg = num_bytes_per_msg / sizeof(int4);
  EP_DEVICE_ASSERT(num_bytes_per_msg % sizeof(int4) == 0);

  // Sending phase
  if ((phases & LOW_LATENCY_SEND_PHASE) == 0) goto LOW_LATENCY_DISPATCH_RECV;

  // Expert counts
  __shared__ int shared_num_tokens_sent_per_expert[kNumWarpGroups];

  // There are 2 kinds of warps in this part:
  // 1. The first-kind warps for FP8 cast and sending top-k tokens
  // 2. The last warp for reading `topk_idx` and count for per-expert
  // information
  if (warp_id < num_warps - 1) {
    constexpr int kNumElemsPerRead = sizeof(int4) / sizeof(hip_bfloat16);
    EP_DEVICE_ASSERT(kHidden % kNumElemsPerRead == 0);
    EP_STATIC_ASSERT(kNumElemsPerRead * 32 % kNumPerChannels == 0,
                     "Invalid vectorization");
    const auto num_threads = (num_warps - 1) * 32;
    const size_t hidden_bf16_int4 = kHidden / kNumElemsPerRead;

    for (int token_idx = sm_id; token_idx < num_tokens; token_idx += num_sms) {
      const auto x_int4 =
          reinterpret_cast<const int4*>(x) + token_idx * hidden_bf16_int4;

      const auto rdma_x_src_idx = reinterpret_cast<int*>(
          reinterpret_cast<uint8_t*>(rdma_x) + token_idx * num_bytes_per_msg);
      const auto rdma_x_vec = reinterpret_cast<vec_t*>(
          reinterpret_cast<uint8_t*>(rdma_x_src_idx) + sizeof(int4));
      const auto rdma_x_scales = reinterpret_cast<float*>(
          reinterpret_cast<uint8_t*>(rdma_x_vec) + hidden_bytes);

      // Overlap top-k index read and source token index write
      auto dst_expert_idx =
          warp_id < num_topk ? static_cast<int>(__ldg(
                                   topk_idx + token_idx * num_topk + warp_id))
                             : -1;
      thread_id == 0 ? (*rdma_x_src_idx = token_idx) : 0;

// FP8 cast
#pragma unroll
      for (int i = thread_id; i < hidden_bf16_int4; i += num_threads) {
        // Read
        auto int4_value = __ldg(x_int4 + i);

        if (kUseFP8) {
          // Calculate local amax
          auto bf16_values = reinterpret_cast<hip_bfloat16*>(&int4_value);
          float fp32_values[kNumElemsPerRead];
          float amax = kFP8Margin, scale, scale_inv;
#pragma unroll
          for (int j = 0; j < kNumElemsPerRead; ++j) {
            fp32_values[j] = static_cast<float>(bf16_values[j]);
            amax = fmaxf(amax, fabsf(fp32_values[j]));
          }

          // Reduce amax and scale
          EP_STATIC_ASSERT(kNumElemsPerRead * 32 / kNumPerChannels == 2,
                           "Invalid vectorization");
          amax = half_warp_reduce_max(amax), scale = kFP8Amax / amax,
          scale_inv = amax * kFP8AmaxInv;
          if (lane_id == 0 || lane_id == 16)
            rdma_x_scales[i * kNumElemsPerRead / 128] = scale_inv;

          // Cast into send buffer
          vec_t int2_value;
          auto fp8x2_values =
              reinterpret_cast<__hip_fp8x2_storage_t*>(&int2_value);
#pragma unroll
          for (int j = 0; j < kNumElemsPerRead; j += 2) {
            float2 fp32x2 = {fp32_values[j] * scale,
                             fp32_values[j + 1] * scale};
            fp8x2_values[j / 2] =
                __hip_cvt_float2_to_fp8x2(fp32x2, __HIP_SATFINITE, __HIP_E4M3_FNUZ);
          }
          rdma_x_vec[i] = int2_value;
        } else {
          // Reinterpret-cast is for C++14 compatibility
          rdma_x_vec[i] = *reinterpret_cast<vec_t*>(&int4_value);
        }
      }
      asm volatile("bar.sync 1, %0;" ::"r"(num_threads));

      // Issue IBGDA sends
      if (dst_expert_idx >= 0) {
        int slot_idx =
            lane_id == 0
                ? atomicAdd(atomic_counter_per_expert + dst_expert_idx, 1)
                : 0;
        slot_idx = __shfl_sync(0xffffffff, slot_idx, 0);
        const auto dst_rank = dst_expert_idx / num_local_experts;
        const auto dst_expert_local_idx = dst_expert_idx % num_local_experts;
        const auto src_ptr = reinterpret_cast<uint64_t>(rdma_x_src_idx);
        const auto dst_ptr =
            reinterpret_cast<uint64_t>(rdma_recv_x) +
            dst_expert_local_idx * num_ranks *
                num_max_dispatch_tokens_per_rank * num_bytes_per_msg +
            rank * num_max_dispatch_tokens_per_rank * num_bytes_per_msg +
            slot_idx * num_bytes_per_msg;
        if (dst_rank != rank) {
          nvshmemi_ibgda_put_nbi_warp(dst_ptr,
                                      src_ptr,
                                      num_bytes_per_msg,
                                      dst_rank,
                                      dst_expert_local_idx,
                                      lane_id,
                                      slot_idx);
        } else {
          // NOTES: only 2 load iterations for 7K hidden with 8 unrolls
          const auto* src_int4_ptr = reinterpret_cast<const int4*>(src_ptr);
          const auto* dst_int4_ptr = reinterpret_cast<int4*>(dst_ptr);
          UNROLLED_WARP_COPY(8,
                             lane_id,
                             num_int4_per_msg,
                             dst_int4_ptr,
                             src_int4_ptr,
                             ld_nc_global,
                             st_na_global);
        }

        // Increase counter after finishing
        __syncwarp();
        lane_id == 0 ? atomic_add_release_global(
                           atomic_finish_counter_per_expert + dst_expert_idx, 1)
                     : 0;
      }
    }
  } else if (warp_id == num_warps - 1) {
    EP_DEVICE_ASSERT(num_sms > 1);
    if (sm_id == 0) {
      // The first SM is also responsible for checking QPs
      EP_DEVICE_ASSERT(ibgda_get_state()->num_rc_per_pe == num_local_experts);

// The first SM is also responsible for cleaning the next buffer
#pragma unroll
      for (int i = lane_id; i < num_next_clean_int; i += 32) next_clean[i] = 0;

      // Notify before executing `int_p`
      __syncwarp();
#pragma unroll
      for (int i = lane_id; i < num_experts; i += 32)
        atomic_add_release_global(atomic_finish_counter_per_expert + i,
                                  FINISHED_SUM_TAG);
    }

    // This SM should be responsible for some destination experts, read
    // `topk_idx` for them
    int expert_count[kNumWarpGroups] = {0};
    const auto expert_begin_idx = sm_id * kNumWarpGroups;
    const auto expert_end_idx =
        min(expert_begin_idx + kNumWarpGroups, num_experts);

// Per lane count
#pragma unroll 8
    for (int i = lane_id; i < num_tokens * num_topk; i += 32) {
      auto idx = static_cast<int>(__ldg(topk_idx + i));
      if (idx >= expert_begin_idx && idx < expert_end_idx)
        expert_count[idx - expert_begin_idx]++;
    }

// Warp reduce
#pragma unroll
    for (int i = expert_begin_idx; i < expert_end_idx; ++i) {
      auto sum = warp_reduce_sum(expert_count[i - expert_begin_idx]);
      if (lane_id == 0) {
        shared_num_tokens_sent_per_expert[i - expert_begin_idx] = sum;
        atomic_add_release_global(atomic_finish_counter_per_expert + i,
                                  FINISHED_SUM_TAG - sum);
      }
    }
  }
  __syncthreads();

  // Issue count sends
  if (responsible_expert_idx < num_experts && sub_warp_id == 0 &&
      lane_id == 0) {
    const auto dst_rank = responsible_expert_idx / num_local_experts;
    const auto dst_expert_local_idx =
        responsible_expert_idx % num_local_experts;
    const auto num_tokens_sent =
        shared_num_tokens_sent_per_expert[responsible_expert_idx -
                                          sm_id * kNumWarpGroups];

    // Wait local sends issued and send expert counts
    while (ld_acquire_global(atomic_finish_counter_per_expert +
                             responsible_expert_idx) != FINISHED_SUM_TAG * 2) {
    }
    if (dst_rank != rank) {
      nvshmemi_ibgda_amo_nonfetch_add(
          rdma_recv_count + dst_expert_local_idx * num_ranks + rank,
          -num_tokens_sent - 1,
          dst_rank,
          dst_expert_local_idx);
    } else {
      st_na_release(rdma_recv_count + dst_expert_local_idx * num_ranks + rank,
                    -num_tokens_sent - 1);
    }

    // Clean workspace for next use
    atomic_counter_per_expert[responsible_expert_idx] = 0;
    atomic_finish_counter_per_expert[responsible_expert_idx] = 0;

    // Clean `packed_recv_count`
    if (dst_rank == 0) packed_recv_count[dst_expert_local_idx] = 0;
  }
  __syncwarp();

// Receiving phase
LOW_LATENCY_DISPATCH_RECV:
  if ((phases & LOW_LATENCY_RECV_PHASE) == 0) return;

  // For send-and-recv kernels, we need a grid sync for making
  // `packed_recv_count` visible
  if (phases & LOW_LATENCY_SEND_PHASE) cg::this_grid().sync();

  // Receiving and packing
  if (responsible_expert_idx < num_experts) {
    const auto src_rank = responsible_expert_idx / num_local_experts;
    const auto local_expert_idx = responsible_expert_idx % num_local_experts;
    const auto rdma_recv_x_uint8 =
        reinterpret_cast<uint8_t*>(rdma_recv_x) +
        local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank *
            num_bytes_per_msg +
        src_rank * num_max_dispatch_tokens_per_rank * num_bytes_per_msg;
    const auto recv_x_int4 = reinterpret_cast<int4*>(packed_recv_x) +
                             local_expert_idx * num_ranks *
                                 num_max_dispatch_tokens_per_rank * hidden_int4;
    const auto recv_x_scales =
        packed_recv_x_scales + local_expert_idx * num_ranks *
                                   num_max_dispatch_tokens_per_rank *
                                   num_scales;
    const auto recv_src_info =
        packed_recv_src_info +
        local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank;
    const auto recv_range =
        packed_recv_layout_range + local_expert_idx * num_ranks;

    // Shared between sub-warps in warp groups
    __shared__ int shared_num_recv_tokens[kNumWarpGroups],
        shared_recv_token_begin_idx[kNumWarpGroups];

    // Wait tokens to arrive
    // NOTES: using sub-warp 1 to overlap with sub-warp 0
    int num_recv_tokens, recv_token_begin_idx;
    EP_STATIC_ASSERT(kNumWarpsPerGroup > 1,
                     "Requires more than one warp per group");
    if (sub_warp_id == 1 && lane_id == 0) {
      while ((num_recv_tokens = ld_acquire_global(
                  rdma_recv_count + local_expert_idx * num_ranks + src_rank)) ==
             0) {
      }
      num_recv_tokens = -num_recv_tokens - 1;
      recv_token_begin_idx =
          atomicAdd(packed_recv_count + local_expert_idx, num_recv_tokens);
      shared_num_recv_tokens[warp_group_id] = num_recv_tokens;
      shared_recv_token_begin_idx[warp_group_id] = recv_token_begin_idx;
      recv_range[src_rank] =
          pack2<int, int64_t>(num_recv_tokens, recv_token_begin_idx);
    }
    asm volatile("bar.sync %0, %1;" ::"r"(warp_group_id + 2),
                 "r"(kNumWarpsPerGroup * 32));
    num_recv_tokens = shared_num_recv_tokens[warp_group_id];
    recv_token_begin_idx = shared_recv_token_begin_idx[warp_group_id];

    // Copy tokens
    EP_DEVICE_ASSERT(num_scales <= 64);
    for (int i = sub_warp_id; i < num_recv_tokens; i += kNumWarpsPerGroup) {
      // Copy source info
      const auto src_src_idx =
          reinterpret_cast<int*>(rdma_recv_x_uint8 + i * num_bytes_per_msg);

      if (lane_id == 0)
        recv_src_info[recv_token_begin_idx + i] = ld_nc_global(src_src_idx);
      __syncwarp();

      // Copy data
      // NOTES: only 2 load iterations for 7K hidden with 7 unrolls
      const auto src_data = reinterpret_cast<int4*>(
          reinterpret_cast<uint8_t*>(src_src_idx) + sizeof(int4));
      const auto dst_data =
          recv_x_int4 + (recv_token_begin_idx + i) * hidden_int4;
      UNROLLED_WARP_COPY(7,
                         lane_id,
                         hidden_int4,
                         dst_data,
                         src_data,
                         ld_nc_global,
                         st_na_global);

      // Copy scales
      if (kUseFP8) {
        const auto src_scales = reinterpret_cast<float*>(
            reinterpret_cast<uint8_t*>(src_data) + hidden_bytes);
        const auto dst_scales =
            reinterpret_cast<float*>(recv_x_scales + recv_token_begin_idx + i);
        const auto scale_stride = num_ranks * num_max_dispatch_tokens_per_rank;
        auto scale_0 =
            lane_id < num_scales ? ld_nc_global(src_scales + lane_id) : 0;
        auto scale_1 = (lane_id + 32) < num_scales
                           ? ld_nc_global(src_scales + lane_id + 32)
                           : 0;
        lane_id < num_scales ? dst_scales[lane_id * scale_stride] = scale_0
                             : 0.0f;
        (lane_id + 32) < num_scales
            ? dst_scales[(lane_id + 32) * scale_stride] = scale_1
            : 0.0f;
      }
    }
  }
}

void dispatch(void* packed_recv_x,
              float* packed_recv_x_scales,
              int* packed_recv_src_info,
              int64_t* packed_recv_layout_range,
              int* packed_recv_count,
              void* rdma_recv_x,
              int* rdma_recv_count,
              void* rdma_x,
              const void* x,
              const int64_t* topk_idx,
              int* next_clean,
              int num_next_clean_int,
              int num_tokens,
              int hidden,
              int num_max_dispatch_tokens_per_rank,
              int num_topk,
              int num_experts,
              int rank,
              int num_ranks,
              bool use_fp8,
              void* workspace,
              hipStream_t stream,
              int phases) {
  constexpr int kNumMaxTopK = 9;
  constexpr int kNumWarpsPerGroup = 10;
  constexpr int kNumWarpGroups = 3;
  EP_STATIC_ASSERT(kNumMaxTopK + 1 <= kNumWarpGroups * kNumWarpsPerGroup,
                   "Too many top-k selections");

  const auto num_warps = kNumWarpGroups * kNumWarpsPerGroup;
  const auto num_sms = cell_div(num_experts, kNumWarpGroups);
  EP_HOST_ASSERT(num_topk <= kNumMaxTopK);
  EP_HOST_ASSERT(cell_div(static_cast<int>(hidden * 2 / sizeof(int4)),
                          32 * (num_warps - 1)) <= 2);

  // Workspace checks
  auto atomic_counter_per_expert = reinterpret_cast<int*>(workspace);
  auto atomic_finish_counter_per_expert =
      atomic_counter_per_expert + num_experts;
  EP_HOST_ASSERT(num_experts * sizeof(int) * 2 <= NUM_WORKSPACE_BYTES);

#define DISPATCH_LAUNCH_CASE(hidden)                                          \
  {                                                                           \
    auto dispatch_func =                                                      \
        use_fp8 ? dispatch<true, kNumWarpGroups, kNumWarpsPerGroup, hidden>   \
                : dispatch<false, kNumWarpGroups, kNumWarpsPerGroup, hidden>; \
    LAUNCH_KERNEL(&cfg,                                                       \
                  dispatch_func,                                              \
                  packed_recv_x,                                              \
                  packed_recv_x_scales,                                       \
                  packed_recv_src_info,                                       \
                  packed_recv_layout_range,                                   \
                  packed_recv_count,                                          \
                  rdma_recv_x,                                                \
                  rdma_recv_count,                                            \
                  rdma_x,                                                     \
                  x,                                                          \
                  topk_idx,                                                   \
                  atomic_counter_per_expert,                                  \
                  atomic_finish_counter_per_expert,                           \
                  next_clean,                                                 \
                  num_next_clean_int,                                         \
                  num_tokens,                                                 \
                  num_max_dispatch_tokens_per_rank,                           \
                  num_topk,                                                   \
                  num_experts,                                                \
                  rank,                                                       \
                  num_ranks,                                                  \
                  phases);                                                    \
  }                                                                           \
  break

  SETUP_LAUNCH_CONFIG(num_sms, num_warps * 32, stream);
  SWITCH_HIDDEN(DISPATCH_LAUNCH_CASE);
#undef DISPATCH_LAUNCH_CASE
}

template <int kNumWarpGroups,
          int kNumWarpsPerGroup,
          int kHidden,
          int kNumMaxTopk>
__global__ __launch_bounds__(
    kNumWarpGroups* kNumWarpsPerGroup * 32,
    1) void combine(void* combined_x,
                    void* rdma_recv_x,
                    int* rdma_recv_flag,
                    void* rdma_send_x,
                    const void* x,
                    const int64_t* topk_idx,
                    const float* topk_weights,
                    const int* src_info,
                    const int64_t* layout_range,
                    int* next_clean,
                    int num_next_clean_int,
                    int* atomic_clean_flag,
                    int num_combined_tokens,
                    int hidden,
                    int num_topk,
                    int num_max_dispatch_tokens_per_rank,
                    int num_experts,
                    int rank,
                    int num_ranks,
                    int phases,
                    bool zero_copy) {
  const auto sm_id = static_cast<int>(blockIdx.x);
  const auto num_sms = static_cast<int>(gridDim.x);
  const auto thread_id = static_cast<int>(threadIdx.x);
  const auto num_threads = static_cast<int>(blockDim.x);
  const auto warp_id = thread_id / 32, lane_id = get_lane_id();
  const auto num_local_experts = num_experts / num_ranks;
  const auto warp_group_id = warp_id / kNumWarpsPerGroup;
  const auto sub_warp_id = warp_id % kNumWarpsPerGroup;
  const auto responsible_expert_idx = sm_id * kNumWarpGroups + warp_group_id;

  // Data type staffs
  constexpr int kNumElemsPerInt4 = sizeof(int4) / sizeof(hip_bfloat16);
  const size_t hidden_bf16_int4 = kHidden / kNumElemsPerInt4;

  // Message package
  // BF16 mode: always use BF16 for hidden data (ignoring the extra flag slot)
  constexpr size_t num_bytes_per_slot =
      sizeof(int4) + kHidden * sizeof(hip_bfloat16);
  EP_STATIC_ASSERT(num_bytes_per_slot % sizeof(int4) == 0,
                   "Invalid vectorization");

  // Sending phase
  if ((phases & LOW_LATENCY_SEND_PHASE) == 0) goto LOW_LATENCY_COMBINE_RECV;

  // Clean up next buffer
  if (sm_id == 0 && warp_group_id == 0 && sub_warp_id == 0) {
#pragma unroll
    for (int i = lane_id; i < num_next_clean_int; i += 32) next_clean[i] = 0;

    // Notify before executing `int_p`
    __syncwarp();
    if (lane_id == 0) atomic_add_release_global(atomic_clean_flag, num_experts);
  }

  // Issue IBGDA sends
  if (responsible_expert_idx < num_experts) {
    const auto dst_rank = responsible_expert_idx / num_local_experts;
    const auto local_expert_idx = responsible_expert_idx % num_local_experts;
    const auto global_expert_idx = rank * num_local_experts + local_expert_idx;
    const auto layout =
        __ldg(layout_range + local_expert_idx * num_ranks +
              dst_rank);  // num_recv_tokens, recv_token_begin_idx

    const auto local_x = reinterpret_cast<const int4*>(x) +
                         local_expert_idx * num_ranks *
                             num_max_dispatch_tokens_per_rank *
                             hidden_bf16_int4;
    const auto local_src_info = src_info + local_expert_idx * num_ranks *
                                               num_max_dispatch_tokens_per_rank;
    const auto rdma_send_x_vec = reinterpret_cast<uint8_t*>(rdma_send_x) +
                                 local_expert_idx * num_ranks *
                                     num_max_dispatch_tokens_per_rank *
                                     num_bytes_per_slot;

    // Unpack layout
    int offset, num_tokens_to_send;
    unpack2(layout, num_tokens_to_send, offset);

    // Issue IBGDA send
    for (int token_idx = offset + sub_warp_id;
         token_idx < offset + num_tokens_to_send;
         token_idx += kNumWarpsPerGroup) {
      const auto x_int4 = local_x + token_idx * hidden_bf16_int4;
      const auto rdma_send_type_row = reinterpret_cast<int*>(
          rdma_send_x_vec + token_idx * num_bytes_per_slot);
      const auto rdma_send_x_vec_row =
          reinterpret_cast<uint8_t*>(rdma_send_type_row + 4);

      // Copy directly to local rank, or copy to buffer and issue RDMA
      auto src_idx = __ldg(local_src_info + token_idx);
      const auto buf_ptr = reinterpret_cast<int64_t>(rdma_send_x_vec_row);
      const auto dst_ptr =
          reinterpret_cast<uint64_t>(rdma_recv_x) +
          (global_expert_idx * num_max_dispatch_tokens_per_rank + src_idx) *
              num_bytes_per_slot +
          sizeof(int4);
      if (dst_rank == rank) {
        const auto dst_int4_ptr = reinterpret_cast<int4*>(dst_ptr);
        UNROLLED_WARP_COPY(7,
                           lane_id,
                           hidden_bf16_int4,
                           dst_int4_ptr,
                           x_int4,
                           ld_nc_global,
                           st_na_global);
      } else {
        const auto buf_int4_ptr = reinterpret_cast<int4*>(buf_ptr);
        if (!zero_copy)
          UNROLLED_WARP_COPY(7,
                             lane_id,
                             hidden_bf16_int4,
                             buf_int4_ptr,
                             x_int4,
                             ld_nc_global,
                             st_na_global);
        nvshmemi_ibgda_put_nbi_warp(dst_ptr,
                                    buf_ptr,
                                    hidden * sizeof(hip_bfloat16),
                                    dst_rank,
                                    local_expert_idx,
                                    lane_id,
                                    token_idx - offset);
      }
    }

    // Put finishing flag
    EP_STATIC_ASSERT(kNumWarpsPerGroup > 1,
                     "Requires more than one warp per group");
    asm volatile("bar.sync %0, %1;" ::"r"(warp_group_id + 1),
                 "r"(kNumWarpsPerGroup * 32));
    if (sub_warp_id == 1 && lane_id == 0) {
      while (ld_acquire_global(atomic_clean_flag) == 0) {
      }
      if (dst_rank != rank) {
        nvshmemi_ibgda_amo_nonfetch_add(
            rdma_recv_flag + global_expert_idx, 1, dst_rank, local_expert_idx);
      } else {
        st_na_release(rdma_recv_flag + global_expert_idx, 1);
      }
      atomic_add_release_global(atomic_clean_flag, -1);
    }
    __syncwarp();
  }

// Receiving phase
LOW_LATENCY_COMBINE_RECV:
  if ((phases & LOW_LATENCY_RECV_PHASE) == 0) return;

  // Wait all ranks to arrive and notify PCIe usage
  if (responsible_expert_idx < num_experts) {
    EP_STATIC_ASSERT(kNumWarpsPerGroup > 1,
                     "Invalid number of warps per group");
    if (sub_warp_id == 0 && lane_id == 0)
      while (ld_acquire_global(rdma_recv_flag + responsible_expert_idx) == 0) {
      }
  }
  cg::this_grid().sync();

  // Reduce tokens with FP8 cast
  // EP_DEVICE_ASSERT(num_topk <= 32 && hidden_bf16_int4 <= num_threads);
  EP_STATIC_ASSERT(kHidden % (32 * kNumElemsPerInt4) == 0,
                   "Invalid vectorization");
  for (int g_id = thread_id; g_id < hidden_bf16_int4; g_id += num_threads) {
    for (int token_idx = sm_id; token_idx < num_combined_tokens;
         token_idx += num_sms) {
      // Read top-k indices and weights
      int reg_topk_idx[kNumMaxTopk];
      float reg_topk_weights[kNumMaxTopk];
#pragma unroll
      for (int i = 0; i < num_topk; ++i) {
        reg_topk_idx[i] =
            static_cast<int>(__ldg(topk_idx + token_idx * num_topk + i));
        reg_topk_weights[i] = __ldg(topk_weights + token_idx * num_topk + i);
      }

      float combined_values[kNumElemsPerInt4] = {0.0f};
#pragma unroll
      for (int i = 0; i < num_topk; ++i)
        if (reg_topk_idx[i] >= 0) {
          // Read from sources
          auto rdma_buffer_type = reinterpret_cast<const int*>(
              reinterpret_cast<uint8_t*>(rdma_recv_x) +
              (reg_topk_idx[i] * num_max_dispatch_tokens_per_rank + token_idx) *
                  num_bytes_per_slot);
          auto rdma_buffer_row =
              reinterpret_cast<const uint8_t*>(rdma_buffer_type + 4);

          // Reduce
          auto x_vec = ld_nc_global(
              reinterpret_cast<const int4*>(rdma_buffer_row) + g_id);
          const auto x_bf16 = reinterpret_cast<hip_bfloat16*>(&x_vec);
#pragma unroll
          for (int j = 0; j < kNumElemsPerInt4; ++j)
            combined_values[j] +=
                static_cast<float>(x_bf16[j]) * reg_topk_weights[i];
        }

      // Write results
      int4& combined_int4 = *reinterpret_cast<int4*>(combined_values);
      auto combined_bf16 = reinterpret_cast<hip_bfloat16*>(&combined_values);
#pragma unroll
      for (int j = 0; j < kNumElemsPerInt4; ++j)
        combined_bf16[j] = static_cast<hip_bfloat16>(combined_values[j]);
      (reinterpret_cast<int4*>(combined_x) +
       token_idx * hidden_bf16_int4)[g_id] = combined_int4;
    }
  }
}

void combine(void* combined_x,
             void* rdma_recv_x,
             int* rdma_recv_flag,
             void* rdma_send_x,
             const void* x,
             const int64_t* topk_idx,
             const float* topk_weights,
             const int* src_info,
             const int64_t* layout_range,
             int* next_clean,
             int num_next_clean_int,
             int num_combined_tokens,
             int hidden,
             int num_max_dispatch_tokens_per_rank,
             int num_topk,
             int num_experts,
             int rank,
             int num_ranks,
             void* workspace,
             hipStream_t stream,
             int phases) {
  constexpr int kNumWarpsPerGroup = 10;
  constexpr int kNumWarpGroups = 3;
  constexpr int kNumMaxTopk = 9;

  const auto num_warps = kNumWarpGroups * kNumWarpsPerGroup;
  const auto num_sms = cell_div(num_experts, kNumWarpGroups);

  // Check workspace
  auto atomic_clean_flag = reinterpret_cast<int*>(workspace);
  EP_HOST_ASSERT(sizeof(int) <= NUM_WORKSPACE_BYTES);
  EP_HOST_ASSERT(num_topk <= kNumMaxTopk);

#define COMBINE_LAUNCH_CASE(hidden)                                      \
  {                                                                      \
    auto combine_func =                                                  \
        combine<kNumWarpGroups, kNumWarpsPerGroup, hidden, kNumMaxTopk>; \
    LAUNCH_KERNEL(&cfg,                                                  \
                  combine_func,                                          \
                  combined_x,                                            \
                  rdma_recv_x,                                           \
                  rdma_recv_flag,                                        \
                  rdma_send_x,                                           \
                  x,                                                     \
                  topk_idx,                                              \
                  topk_weights,                                          \
                  src_info,                                              \
                  layout_range,                                          \
                  next_clean,                                            \
                  num_next_clean_int,                                    \
                  atomic_clean_flag,                                     \
                  num_combined_tokens,                                   \
                  hidden,                                                \
                  num_topk,                                              \
                  num_max_dispatch_tokens_per_rank,                      \
                  num_experts,                                           \
                  rank,                                                  \
                  num_ranks,                                             \
                  phases,                                                \
                  false);                                                \
  }                                                                      \
  break

  SETUP_LAUNCH_CONFIG(num_sms, num_warps * 32, stream);
  SWITCH_HIDDEN(COMBINE_LAUNCH_CASE);
#undef COMBINE_LAUNCH_CASE
}

}  // namespace internode_ll

}  // namespace deep_ep
