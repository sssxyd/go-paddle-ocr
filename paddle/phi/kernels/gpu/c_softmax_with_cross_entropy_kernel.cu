#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/platform/collective_helper.h"
#include "paddle/phi/kernels/activation_kernel.h"
#include "paddle/phi/kernels/full_kernel.h"
#include "paddle/phi/kernels/funcs/axis_utils.h"
#include "paddle/phi/kernels/funcs/broadcast_function.h"
#include "paddle/phi/kernels/funcs/cross_entropy.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"
#include "paddle/phi/kernels/funcs/elementwise_functor.h"
#include "paddle/phi/kernels/funcs/math.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/softmax.h"
#include "paddle/phi/kernels/funcs/softmax_impl.h"
#include "paddle/phi/kernels/reduce_max_kernel.h"
#include "paddle/phi/kernels/reduce_sum_kernel.h"
#include "paddle/utils/string/string_helper.h"

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
#include "paddle/phi/core/distributed/nccl_comm_context.h"
#endif

namespace phi {

template <typename Context, typename T>
struct CSoftmaxWithCrossEntropyFunctor {
  void operator()(const Context& dev_ctx,
                  const DenseTensor& logits,
                  const DenseTensor& label,
                  int64_t ignore_index,
                  int rank,
                  int nranks,
                  DenseTensor* softmax,
                  DenseTensor* loss);
};

static constexpr int kNumCUDAThreads = 512;
static constexpr int64_t kNumMaximumNumBlocks = 4096;

static inline int64_t NumBlocks(const int64_t N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaximumNumBlocks);
}

template <typename T, typename IndexT>
__global__ void MaskLabelByIndex(T* predicted_logits,
                                 const T* logit,
                                 const IndexT* label,
                                 const IndexT ignore_index,
                                 const int64_t start_index,
                                 const int64_t end_index,
                                 const int64_t N,
                                 const int64_t D,
                                 const int nranks) {
  CUDA_KERNEL_LOOP_TYPE(i, N, int64_t) {
    auto real_label = label[i];
    PADDLE_ENFORCE(((real_label < D * nranks) && (real_label >= 0)) ||
                       (real_label == ignore_index),
                   "The index is out of bounds, "
                   "please check whether the value of label and "
                   "input meet the class number. It should "
                   "be less than [%ld] or equal to [%ld], but received [%ld]",
                   static_cast<int64_t>(D * nranks),
                   static_cast<int64_t>(ignore_index),
                   static_cast<int64_t>(real_label));

    if (real_label >= start_index && real_label < end_index) {
      predicted_logits[i] = logit[i * D + real_label - start_index];
    }
  }
}

template <typename T, typename IndexT>
__global__ void SoftMaskLabelByIndex(T* predicted_logits,
                                     const T* logit,
                                     const IndexT* label,
                                     const IndexT ignore_index,
                                     const int64_t start_index,
                                     const int64_t end_index,
                                     const int64_t N,
                                     const int64_t D,
                                     const int64_t C,
                                     const int nranks) {
  CUDA_KERNEL_LOOP_TYPE(i, N, int64_t) {
    for (int j = 0; j < C; ++j) {
      auto real_label = label[i * C + j];
      PADDLE_ENFORCE(((real_label < D * nranks) && (real_label >= 0)) ||
                         (real_label == ignore_index),
                     "The index is out of bounds, "
                     "please check whether the value of label and "
                     "input meet the class number. It should "
                     "be less than [%ld] or equal to [%ld], but received [%ld]",
                     static_cast<int64_t>(D * nranks),
                     static_cast<int64_t>(ignore_index),
                     static_cast<int64_t>(real_label));

      if (real_label >= start_index && real_label < end_index) {
        predicted_logits[i * C + j] = logit[i * D + real_label - start_index];
      }
    }
  }
}

template <typename T, typename IndexT>
__global__ void CalculateLoss(T* loss,
                              const T* predict_logits,
                              const T* sum_exp_logits,
                              const IndexT* label,
                              const int64_t ignore_index,
                              const int64_t N) {
  CUDA_KERNEL_LOOP_TYPE(i, N, int64_t) {
    auto real_label = static_cast<int64_t>(label[i]);
    loss[i] = ignore_index == real_label
                  ? static_cast<T>(0)
                  : phi::funcs::TolerableValue<T>()(
                        phi::funcs::TolerableValue<T>()(
                            phi::funcs::real_log(sum_exp_logits[i])) -
                        predict_logits[i]);
  }
}

template <typename T, typename IndexT>
__global__ void CalculateSoftLoss(T* loss,
                                  const T* predict_logits,
                                  const T* sum_exp_logits,
                                  const IndexT* label,
                                  const int64_t ignore_index,
                                  const int64_t N,
                                  const int64_t C) {
  const T prob = static_cast<T>(1.0 / C);

  CUDA_KERNEL_LOOP_TYPE(i, N, int64_t) {
    T tmp_loss = static_cast<T>(0);
    int ignore_num = 0;
    for (int j = 0; j < C; ++j) {
      auto real_label = static_cast<int64_t>(label[i * C + j]);
      tmp_loss += ignore_index == real_label
                      ? static_cast<T>(0)
                      : phi::funcs::TolerableValue<T>()(
                            (phi::funcs::TolerableValue<T>()(
                                 phi::funcs::real_log(sum_exp_logits[i])) -
                             predict_logits[i * C + j]) *
                            prob);
      ignore_num += ignore_index == real_label ? 1 : 0;
    }
    loss[i] = ignore_num > 0 ? static_cast<T>(0) : tmp_loss;
  }
}

template <typename T, typename Context>
void CSoftmaxWithCrossEntropyKernel(const Context& dev_ctx,
                                    const DenseTensor& logits,
                                    const DenseTensor& label,
                                    int64_t ignore_index,
                                    int rank,
                                    int nranks,
                                    DenseTensor* softmax,
                                    DenseTensor* loss) {
  CSoftmaxWithCrossEntropyFunctor<phi::GPUContext, T> functor_;
  functor_(dev_ctx, logits, label, ignore_index, rank, nranks, softmax, loss);
}

template <typename T>
struct CSoftmaxWithCrossEntropyFunctor<phi::GPUContext, T> {
  void operator()(const phi::GPUContext& dev_ctx,
                  const DenseTensor& logits_in,
                  const DenseTensor& label_in,
                  int64_t ignore_index,
                  int rank,
                  int nranks,
                  DenseTensor* softmax,
                  DenseTensor* loss) {
#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
    const phi::DenseTensor* logits = &logits_in;
    const phi::DenseTensor* labels = &label_in;

    gpuStream_t stream = nullptr;
    phi::distributed::NCCLCommContext* comm_ctx = nullptr;

    comm_ctx = static_cast<phi::distributed::NCCLCommContext*>(
        dev_ctx.GetCommContext());
    PADDLE_ENFORCE_NE(comm_ctx,
                      nullptr,
                      common::errors::Unavailable(
                          "NCCLCommContext is nullptr, collective op should "
                          "has ring_id attr."));

    stream = dev_ctx.stream();

    // allocate memory on device.
    dev_ctx.template Alloc<T>(softmax);
    dev_ctx.template Alloc<T>(loss);

    const auto& logits_dims = logits->dims();
    const auto& labels_dims = labels->dims();

    const int axis = logits_dims.size() - 1;
    const int64_t N = phi::funcs::SizeToAxis<int64_t>(axis, logits_dims);
    const int64_t D = phi::funcs::SizeFromAxis<int64_t>(axis, logits_dims);
    const int64_t C = phi::funcs::SizeFromAxis<int64_t>(axis, labels_dims);

    phi::DenseTensor logits_2d, softmax_2d, loss_2d;
    logits_2d.ShareDataWith(*logits).Resize({N, D});
    softmax_2d.ShareDataWith(*softmax).Resize({N, D});
    loss_2d.ShareDataWith(*loss).Resize({N, 1});

    // step 1, obtain logit_max
    phi::DenseTensor logits_max;
    logits_max.Resize({N, 1});
    dev_ctx.template Alloc<T>(&logits_max);

    phi::MaxKernel<T, phi::GPUContext>(
        dev_ctx, logits_2d, {-1}, true, &logits_max);

    comm_ctx->AllReduce(&logits_max, logits_max, ncclMax, stream);

    // step 2, obtain logit - logit_max
    std::vector<const phi::DenseTensor*> inputs = {&logits_2d, &logits_max};
    std::vector<phi::DenseTensor*> outputs = {&softmax_2d};
    phi::funcs::BroadcastKernel<T>(
        dev_ctx, inputs, &outputs, phi::funcs::SubtractFunctor<T>());

    // step 3, obtain predict target
    phi::DenseTensor predicted_logits;
    predicted_logits.Resize({N, 1});
    dev_ctx.template Alloc<T>(&predicted_logits);

    phi::FullKernel<T, phi::GPUContext>(
        dev_ctx,
        common::vectorize(predicted_logits.dims()),
        0,
        predicted_logits.dtype(),
        &predicted_logits);

    const int64_t start_index = rank * D;
    const int64_t end_index = start_index + D;

    int64_t blocks = NumBlocks(N);
    int threads = kNumCUDAThreads;
    const auto& label_type = labels->dtype();

    if (label_type == phi::DataType::INT32) {
      if (C > 1) {
        SoftMaskLabelByIndex<T, int32_t>
            <<<blocks, threads, 0, dev_ctx.stream()>>>(
                predicted_logits.data<T>(),
                softmax_2d.data<T>(),
                labels->data<int32_t>(),
                static_cast<int32_t>(ignore_index),
                start_index,
                end_index,
                N,
                D,
                C,
                nranks);
      } else {
        MaskLabelByIndex<T, int32_t><<<blocks, threads, 0, dev_ctx.stream()>>>(
            predicted_logits.data<T>(),
            softmax_2d.data<T>(),
            labels->data<int32_t>(),
            static_cast<int32_t>(ignore_index),
            start_index,
            end_index,
            N,
            D,
            nranks);
      }
    } else if (label_type == phi::DataType::INT64) {
      if (C > 1) {
        SoftMaskLabelByIndex<T, int64_t>
            <<<blocks, threads, 0, dev_ctx.stream()>>>(
                predicted_logits.data<T>(),
                softmax_2d.data<T>(),
                labels->data<int64_t>(),
                ignore_index,
                start_index,
                end_index,
                N,
                D,
                C,
                nranks);
      } else {
        MaskLabelByIndex<T, int64_t><<<blocks, threads, 0, dev_ctx.stream()>>>(
            predicted_logits.data<T>(),
            softmax_2d.data<T>(),
            labels->data<int64_t>(),
            ignore_index,
            start_index,
            end_index,
            N,
            D,
            nranks);
      }
    }

    dev_ctx.template Alloc<T>(&predicted_logits);
    comm_ctx->AllReduce(&predicted_logits, predicted_logits, ncclSum, stream);

    // step 4, obtain exp(logit)
    phi::ExpKernel<T, phi::GPUContext>(dev_ctx, softmax_2d, &softmax_2d);

    // step 5, obtain sum_exp_logits
    phi::DenseTensor sum_exp_logits;
    sum_exp_logits.Resize({N, 1});
    dev_ctx.template Alloc<T>(&sum_exp_logits);

    phi::SumKernel<T, phi::GPUContext>(
        dev_ctx, softmax_2d, {-1}, softmax_2d.dtype(), true, &sum_exp_logits);

    comm_ctx->AllReduce(&sum_exp_logits, sum_exp_logits, ncclSum, stream);

    if (label_type == phi::DataType::INT32) {
      if (C > 1) {
        CalculateSoftLoss<T, int32_t><<<blocks, threads, 0, dev_ctx.stream()>>>(
            loss_2d.data<T>(),
            predicted_logits.data<T>(),
            sum_exp_logits.data<T>(),
            labels->data<int32_t>(),
            ignore_index,
            N,
            C);
      } else {
        CalculateLoss<T, int32_t><<<blocks, threads, 0, dev_ctx.stream()>>>(
            loss_2d.data<T>(),
            predicted_logits.data<T>(),
            sum_exp_logits.data<T>(),
            labels->data<int32_t>(),
            ignore_index,
            N);
      }

    } else {
      if (C > 1) {
        CalculateSoftLoss<T, int64_t><<<blocks, threads, 0, dev_ctx.stream()>>>(
            loss_2d.data<T>(),
            predicted_logits.data<T>(),
            sum_exp_logits.data<T>(),
            labels->data<int64_t>(),
            ignore_index,
            N,
            C);
      } else {
        CalculateLoss<T, int64_t><<<blocks, threads, 0, dev_ctx.stream()>>>(
            loss_2d.data<T>(),
            predicted_logits.data<T>(),
            sum_exp_logits.data<T>(),
            labels->data<int64_t>(),
            ignore_index,
            N);
      }
    }

    phi::ReciprocalKernel<T, phi::GPUContext>(
        dev_ctx, sum_exp_logits, &sum_exp_logits);

    inputs = std::vector<const phi::DenseTensor*>{&softmax_2d, &sum_exp_logits};
    outputs = std::vector<phi::DenseTensor*>{&softmax_2d};
    phi::funcs::BroadcastKernel<T>(
        dev_ctx, inputs, &outputs, phi::funcs::MultiplyFunctor<T>());
#endif
  }
};

}  // namespace phi

PD_REGISTER_KERNEL(c_softmax_with_cross_entropy,
                   GPU,
                   ALL_LAYOUT,
                   phi::CSoftmaxWithCrossEntropyKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
